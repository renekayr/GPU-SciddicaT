#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "util.hpp"

// ----------------------------------------------------------------------------
// I/O parameters used to index argv[]
// ----------------------------------------------------------------------------
#define HEADER_PATH_ID 1
#define DEM_PATH_ID 2
#define SOURCE_PATH_ID 3
#define OUTPUT_PATH_ID 4
#define STEPS_ID 5

// ----------------------------------------------------------------------------
// Simulation parameters
// ----------------------------------------------------------------------------
#define P_R 0.5
#define P_EPSILON 0.001
#define ADJACENT_CELLS 4
#define STRLEN 256

// ----------------------------------------------------------------------------
// Tiled Halo Cell algorithm parameters
// ----------------------------------------------------------------------------
  // Von Neumann Neighborhood
#define MASK_WIDTH 3
  // Tile size can be dynamically calculated by using tile_width = 1 - mask_width - (pow(max_shared_memory, 2) / pow(sizeof(datatype), 2))
  // max_shared_memory can be queried from the CUDA API at runtime
  // This formula is derived by solving the following equation for for tile_width:
  // max_shared_memory = (mask_width + tile_width - 1)^2 * sizeof(datatype)
  // Else, an arbitrary or estimated amount that does not surpass the GPU's capacity is chosen
#define TILE_WIDTH_FLOWSCOMPUTATION 27
#define TILE_WIDTH_WIDTHUPDATE 32

// ----------------------------------------------------------------------------
// Read/Write access macros linearizing single/multy layer buffer 2D indices
// ----------------------------------------------------------------------------
#define SET(M, columns, i, j, value) ((M)[(((i) * (columns)) + (j))] = (value))
#define GET(M, columns, i, j) (M[(((i) * (columns)) + (j))])
#define BUF_SET(M, rows, columns, n, i, j, value) ( (M)[( ((n)*(rows)*(columns)) + ((i)*(columns)) + (j) )] = (value) )
#define BUF_GET(M, rows, columns, n, i, j) ( M[( ((n)*(rows)*(columns)) + ((i)*(columns)) + (j) )] )

// ----------------------------------------------------------------------------
// I/O functions
// ----------------------------------------------------------------------------
void readHeaderInfo(char* path, int &nrows, int &ncols, double &nodata)
{
  FILE* f;
  
  if ((f = fopen(path, "r")) == 0) {
    printf("%s configuration header file not found\n", path);
    exit(0);
  }

  //Read the header
  char str[STRLEN];
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); ncols = atoi(str);      //ncols
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); nrows = atoi(str);      //nrows
  fscanf(f,"%s",&str); fscanf(f,"%s",&str);                         //xllcorner
  fscanf(f,"%s",&str); fscanf(f,"%s",&str);                         //yllcorner
  fscanf(f,"%s",&str); fscanf(f,"%s",&str);                         //cellsize
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); nodata = atof(str);     //NODATA_value 
}

bool loadGrid2D(double *M, int rows, int columns, char *path)
{
  FILE *f = fopen(path, "r");

  if (!f) {
    printf("%s grid file not found\n", path);
    exit(0);
  }

  char str[STRLEN];
  for (int i = 0; i < rows; ++i)
    for (int j = 0; j < columns; ++j)
    {
      fscanf(f, "%s", str);
      SET(M, columns, i, j, atof(str));
    }

  fclose(f);

  return true;
}

bool saveGrid2Dr(double *M, int rows, int columns, char *path)
{
  FILE *f;
  f = fopen(path, "w");

  if (!f)
    return false;

  char str[STRLEN];
  for (int i = 0; i < rows; ++i)
  {
    for (int j = 0; j < columns; ++j)
    {
      sprintf(str, "%f ", GET(M, columns, i, j));
      fprintf(f, "%s ", str);
    }
    fprintf(f, "\n");
  }

  fclose(f);

  return true;
}

double* addLayer2D(int rows, int columns)
{
  double *tmp;
  checkError(hipMallocManaged(&tmp, sizeof(double) * rows * columns), __LINE__, "error allocating memory");

  if (!tmp)
    return NULL;
  return tmp;
}

// ----------------------------------------------------------------------------
// init kernel, called once before the simulation loop
// ----------------------------------------------------------------------------
__global__ void sciddicaTSimulationInitKernel(int r, int c, double *Sz, double *Sh)
{
  int col_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int row_idx = threadIdx.y + blockDim.y * blockIdx.y;
  int col_stride = blockDim.x * gridDim.x;
  int row_stride = blockDim.y * gridDim.y;

  double z, h;

  for (int row = row_idx + 1; row < r - 1; row += row_stride)
    for (int col = col_idx + 1; col < c - 1; col += col_stride) {
      h = GET(Sh, c, row, col);

      if (h > 0.0) {
        z = GET(Sz, c, row, col);
        SET(Sz, c, row, col, z - h);
      }
    }
}

// ----------------------------------------------------------------------------
// computing kernels, aka elementary processes in the XCA terminology
// ----------------------------------------------------------------------------
__global__ void sciddicaTResetFlowsKernel(int r, int c, double nodata, double* Sf)
{
  int col_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int row_idx = threadIdx.y + blockDim.y * blockIdx.y;
  int col_stride = blockDim.x * gridDim.x;
  int row_stride = blockDim.y * gridDim.y;

  for (int row = row_idx + 1; row < r - 1; row += row_stride)
    for (int col = col_idx + 1; col < c - 1; col += col_stride)
      for(int cnt = 0; cnt <= MASK_WIDTH; ++cnt)
        BUF_SET(Sf, r, c, cnt, row, col, nodata);
}

__global__ void sciddicaTFlowsComputationCachingKernel(int r, int c, double nodata, int *Xi, int *Xj, double *Sz, double *Sh, double *Sf, double p_r, double p_epsilon)
{
  int col_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int row_idx = threadIdx.y + blockDim.y * blockIdx.y;

  int tile_start_x = blockIdx.x * blockDim.x;
  int next_tile_start_x = ((blockIdx.x + 1) * blockDim.x);
  int tile_start_y = blockIdx.y * blockDim.y;
  int next_tile_start_y = ((blockIdx.y + 1) * blockDim.y);

  __shared__ double Sz_ds[TILE_WIDTH_FLOWSCOMPUTATION][TILE_WIDTH_FLOWSCOMPUTATION];
  __shared__ double Sh_ds[TILE_WIDTH_FLOWSCOMPUTATION][TILE_WIDTH_FLOWSCOMPUTATION];

  bool eliminated_cells[5] = {false, false, false, false, false};
  bool again;
  int cells_count;
  double average;
  double m;
  double u[5];
  int n;
  double z = 0, h = 0;

  Sz_ds[threadIdx.y][threadIdx.x] = GET(Sz, c, row_idx, col_idx);
  Sh_ds[threadIdx.y][threadIdx.x] = GET(Sh, c, row_idx, col_idx);
  __syncthreads();

  if (col_idx > 0 && col_idx < c - 1 && row_idx > 0 && row_idx < r - 1) {
    m = Sh_ds[threadIdx.y][threadIdx.x] - p_epsilon;
    u[0] = Sz_ds[threadIdx.y][threadIdx.x] + p_epsilon;

    for (int cnt = 0; cnt <= MASK_WIDTH; ++cnt) {
      int n_index_y = row_idx + Xi[cnt+1];
      int n_index_x = col_idx + Xj[cnt+1];

      if ((n_index_x >= 0) && (n_index_x < c) && (n_index_y >= 0) && (n_index_y < r)) {
        if ((n_index_x >= tile_start_x) && (n_index_x < next_tile_start_x) && (n_index_y >= tile_start_y) && (n_index_y < next_tile_start_y)) {
          z = Sz_ds[threadIdx.y + Xi[cnt+1]][threadIdx.x + Xj[cnt+1]];
          h = Sh_ds[threadIdx.y + Xi[cnt+1]][threadIdx.x + Xj[cnt+1]];
        }
        else {
          z = GET(Sz, c, n_index_y, n_index_x);
          h = GET(Sh, c, n_index_y, n_index_x);
        }
        u[cnt+1] = z + h;
      }
    }

    do {
      again = false;
      average = m;
      cells_count = 0;

      for (n = 0; n < 5; ++n)
        if (!eliminated_cells[n]) {
          average += u[n];
          ++cells_count;
        }

      if (cells_count != 0)
        average /= cells_count;

      for (n = 0; n < 5; ++n)
        if ((average <= u[n]) && (!eliminated_cells[n])) {
          eliminated_cells[n] = true;
          again = true;
        }
    } while (again);

    for(int cnt = 0; cnt < 4; ++cnt)
      if (!eliminated_cells[cnt+1])
        BUF_SET(Sf, r, c, cnt, row_idx, col_idx, (average - u[cnt+1]) * p_r);
  }
}

__global__ void sciddicaTWidthUpdateCachingKernel(int r, int c, double nodata, int *Xi, int *Xj, double *Sz, double *Sh, double *Sf)
{
  int col_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int row_idx = threadIdx.y + blockDim.y * blockIdx.y;

  int tile_start_x = blockIdx.x * blockDim.x;
  int next_tile_start_x = ((blockIdx.x + 1) * blockDim.x);
  int tile_start_y = blockIdx.y * blockDim.y;
  int next_tile_start_y = ((blockIdx.y + 1) * blockDim.y);

  __shared__ double Sf_ds[TILE_WIDTH_WIDTHUPDATE * ADJACENT_CELLS][TILE_WIDTH_WIDTHUPDATE];

  for(int cnt = 0; cnt < 4; ++cnt)
    Sf_ds[threadIdx.y + cnt * TILE_WIDTH_WIDTHUPDATE][threadIdx.x] = BUF_GET(Sf, r, c, cnt, row_idx, col_idx);
  __syncthreads();

  if (col_idx > 0 && col_idx < c - 1 && row_idx > 0 && row_idx < r - 1) {
    double h_next = GET(Sh, c, row_idx, col_idx);

    for (int cnt = 0; cnt <= MASK_WIDTH; ++cnt) {
      int n_index_x = col_idx + Xj[cnt+1];
      int n_index_y = row_idx + Xi[cnt+1];
      if ((n_index_x >= 0) && (n_index_x < c) && (n_index_y >= 0) && (n_index_y < r))
        if ((n_index_x >= tile_start_x) && (n_index_x < next_tile_start_x) && (n_index_y >= tile_start_y) && (n_index_y < next_tile_start_y))
          h_next += Sf_ds[threadIdx.y + Xi[cnt+1] + (MASK_WIDTH - cnt) * TILE_WIDTH_WIDTHUPDATE][threadIdx.x + Xj[cnt+1]]
                    - Sf_ds[threadIdx.y + cnt * TILE_WIDTH_WIDTHUPDATE][threadIdx.x];
        else
          h_next += BUF_GET(Sf, r, c, (MASK_WIDTH - cnt), n_index_y, n_index_x)
                    - BUF_GET(Sf, r, c, cnt, row_idx, col_idx);
    }
    SET(Sh, c, row_idx, col_idx, h_next);
  }
}

// ----------------------------------------------------------------------------
// Function main()
// ----------------------------------------------------------------------------
int main(int argc, char **argv)
{
  int rows, cols;
  double nodata;
  readHeaderInfo(argv[HEADER_PATH_ID], rows, cols, nodata);

  int r = rows;                     // r: grid rows
  int c = cols;                     // c: grid columns
  double *Sz;                       // Sz: substate (grid) containing cells' altitude a.s.l.
  double *Sh;                       // Sh: substate (grid) containing cells' flow thickness
  double *Sf;                       // Sf: 4 substates containing the flows towards the 4 neighbors
  int* Xi;                          // Xj: von Neuman neighborhood row coordinates (see below)
  int* Xj;                          // Xj: von Neuman neighborhood col coordinates (see below)
  double p_r = P_R;                 // p_r: minimization algorithm outflows dumping factor
  double p_epsilon = P_EPSILON;     // p_epsilon: frictional parameter threshold
  int steps = atoi(argv[STEPS_ID]); //steps: simulation steps

  // The adopted von Neuman neighborhood
  // Format: flow_index:cell_label:(row_index,col_index)
  //
  //   cell_label in [0,1,2,3,4]: label assigned to each cell in the neighborhood
  //   flow_index in   [0,1,2,3]: outgoing flow indices in Sf from cell 0 to the others
  //       (row_index,col_index): 2D relative indices of the cells
  //
  //               |0:1:(-1, 0)|
  //   |1:2:( 0,-1)| :0:( 0, 0)|2:3:( 0, 1)|
  //               |3:4:( 1, 0)|
  //

  // printf("Allocating memory...\n");
  Sz = addLayer2D(r, c);                  // Allocates the Sz substate grid
  Sh = addLayer2D(r, c);                  // Allocates the Sh substate grid
  Sf = addLayer2D(ADJACENT_CELLS * r, c); // Allocates the Sf substates grid, having one layer for each adjacent cell
  checkError(hipMallocManaged(&Xi, sizeof(int) * 5), __LINE__, "error allocating memory for Xi");
  Xi[0] = 0; Xi[1] = -1; Xi[2] = 0;  Xi[3] = 0; Xi[4] = 1;
  checkError(hipMallocManaged(&Xj, sizeof(int) * 5), __LINE__, "error allocating memory for Xj");
  Xj[0] = 0; Xj[1] = 0;  Xj[2] = -1; Xj[3] = 1; Xj[4] = 0;

  // printf("Loading data from file...\n");
  loadGrid2D(Sz, r, c, argv[DEM_PATH_ID]);    // Load Sz from file
  loadGrid2D(Sh, r, c, argv[SOURCE_PATH_ID]); // Load Sh from file

  int n = rows * cols;
  int dim_x = 32;
  int dim_y = 32;
  dim3 block_size(dim_x, dim_y, 1);
  dim3 grid_size(ceil(sqrt(n / (dim_x * dim_y))), ceil(sqrt(n / (dim_x * dim_y))), 1);

  printf("\n");
  printf("Problem size is %d elements\n", n);
  printf("\n");

  dim3 tiled_block_size_flowscomputation(TILE_WIDTH_FLOWSCOMPUTATION, TILE_WIDTH_FLOWSCOMPUTATION, 1);
  dim3 tiled_grid_size_flowscomputation(ceil(sqrt(n / (TILE_WIDTH_FLOWSCOMPUTATION * TILE_WIDTH_FLOWSCOMPUTATION))),
                                        ceil(sqrt(n / (TILE_WIDTH_FLOWSCOMPUTATION * TILE_WIDTH_FLOWSCOMPUTATION))), 1);

  printf("\n");
  printf("*---------- FlowsComputation ----------*\n");
  printf("Tile width is %d\n", TILE_WIDTH_FLOWSCOMPUTATION);
  printf("Tiled block dimensions are %d, %d, %d\n", tiled_block_size_flowscomputation.x, tiled_block_size_flowscomputation.y, tiled_block_size_flowscomputation.z);
  printf("Tiled grid dimensions are %d, %d, %d\n", tiled_grid_size_flowscomputation.x, tiled_grid_size_flowscomputation.y, tiled_grid_size_flowscomputation.z);
  printf("Total blocks in tiled grid are: %d\n", tiled_grid_size_flowscomputation.x * tiled_grid_size_flowscomputation.y * tiled_grid_size_flowscomputation.z);
  printf("Total tiled grid threads are: %d\n", tiled_block_size_flowscomputation.x * tiled_block_size_flowscomputation.y * tiled_block_size_flowscomputation.z * tiled_grid_size_flowscomputation.x * tiled_grid_size_flowscomputation.y * tiled_grid_size_flowscomputation.z);
  printf("Threads only involved in output: %d\n", TILE_WIDTH_FLOWSCOMPUTATION * TILE_WIDTH_FLOWSCOMPUTATION * tiled_grid_size_flowscomputation.x * tiled_grid_size_flowscomputation.y * tiled_grid_size_flowscomputation.z);
  printf("One double precision buffer requires %lld bytes of shared memory\n", TILE_WIDTH_FLOWSCOMPUTATION * TILE_WIDTH_FLOWSCOMPUTATION * sizeof(double));
  printf("\n");

  dim3 tiled_block_size_widthupdate(TILE_WIDTH_WIDTHUPDATE, TILE_WIDTH_WIDTHUPDATE, 1);
  dim3 tiled_grid_size_widthupdate(ceil(sqrt(n / (TILE_WIDTH_WIDTHUPDATE * TILE_WIDTH_WIDTHUPDATE))),
                                        ceil(sqrt(n / (TILE_WIDTH_WIDTHUPDATE * TILE_WIDTH_WIDTHUPDATE))), 1);

  printf("\n");
  printf("*---------- WidthUpdate ----------*\n");
  printf("Tile width is %d\n", TILE_WIDTH_WIDTHUPDATE);
  printf("Tiled block dimensions are %d, %d, %d\n", tiled_block_size_widthupdate.x, tiled_block_size_widthupdate.y, tiled_block_size_widthupdate.z);
  printf("Tiled grid dimensions are %d, %d, %d\n", tiled_grid_size_widthupdate.x, tiled_grid_size_widthupdate.y, tiled_grid_size_widthupdate.z);
  printf("Total blocks in tiled grid are: %d\n", tiled_grid_size_widthupdate.x * tiled_grid_size_widthupdate.y * tiled_grid_size_widthupdate.z);
  printf("Total tiled grid threads are: %d\n", tiled_block_size_widthupdate.x * tiled_block_size_widthupdate.y * tiled_block_size_widthupdate.z * tiled_grid_size_widthupdate.x * tiled_grid_size_widthupdate.y * tiled_grid_size_widthupdate.z);
  printf("Threads only involved in output: %d\n", TILE_WIDTH_WIDTHUPDATE * TILE_WIDTH_WIDTHUPDATE * tiled_grid_size_widthupdate.x * tiled_grid_size_widthupdate.y * tiled_grid_size_widthupdate.z);
  printf("One double precision buffer requires %lld bytes of shared memory\n", TILE_WIDTH_WIDTHUPDATE * TILE_WIDTH_WIDTHUPDATE * sizeof(double));
  printf("\n");

  //printf("Initializing...\n");
  sciddicaTSimulationInitKernel<<<grid_size, block_size>>>(r, c, Sz, Sh);
  checkError(__LINE__, "error executing sciddicaTSimulationInitKernel");
  checkError(hipDeviceSynchronize(), __LINE__, "error syncing after sciddicaTSimulationInitKernel");

  // int loops = 100;  // TEST
  printf("Running the simulation for %d steps...\n", steps);
  // printf("... and %d times, determining the best time.\n", loops);
  // double best_time = 0.0;
  // for(int loop = 0; loop < loops; ++loop) {
    util::Timer cl_timer;
    for (int s = 0; s < steps; ++s) {
      //printf("step %d\n", s+1);

      sciddicaTResetFlowsKernel<<<grid_size, block_size>>>(r, c, nodata, Sf);
      checkError(__LINE__, "error executing sciddicaTSimulationInitKernel");
      checkError(hipDeviceSynchronize(), __LINE__, "error syncing after sciddicaTResetFlowsKernel");

      sciddicaTFlowsComputationCachingKernel<<<tiled_grid_size_flowscomputation, tiled_block_size_flowscomputation>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf, p_r, p_epsilon);
      checkError(__LINE__, "error executing sciddicaTFlowsComputationCachingKernel");
      checkError(hipDeviceSynchronize(), __LINE__, "error syncing after sciddicaTFlowsComputationCachingKernel");

      sciddicaTWidthUpdateCachingKernel<<<tiled_grid_size_widthupdate, tiled_block_size_widthupdate>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf);
      checkError(__LINE__, "error executing sciddicaTWidthUpdateCachingKernel");
      checkError(hipDeviceSynchronize(), __LINE__, "error syncing after sciddicaTWidthUpdateCachingKernel");
    }
    double cl_time = static_cast<double>(cl_timer.getTimeMilliseconds()) / 1000.0;
    // printf("[%d] ", loop);
    printf("Elapsed time: %lf [s]\n", cl_time);
    // if(cl_time < best_time || loop == 0) {
    //   best_time = cl_time;
    // }
  // }
  // printf("Best time: %lf [s]\n", best_time);

  saveGrid2Dr(Sh, r, c, argv[OUTPUT_PATH_ID]);

  //printf("Releasing memory...\n");
  checkError(hipFree(Sz), __LINE__, "error deallocating memory for Sz");
  checkError(hipFree(Sh), __LINE__, "error deallocating memory for Sh");
  checkError(hipFree(Sf), __LINE__, "error deallocating memory for Sf");
  checkError(hipFree(Xi), __LINE__, "error deallocating memory for Xi");
  checkError(hipFree(Xj), __LINE__, "error deallocating memory for Xj");

  return 0;
}
