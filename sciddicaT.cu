#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "util.hpp"

// ----------------------------------------------------------------------------
// I/O parameters used to index argv[]
// ----------------------------------------------------------------------------
#define HEADER_PATH_ID 1
#define DEM_PATH_ID 2
#define SOURCE_PATH_ID 3
#define OUTPUT_PATH_ID 4
#define STEPS_ID 5

// ----------------------------------------------------------------------------
// Simulation parameters
// ----------------------------------------------------------------------------
#define P_R 0.5
#define P_EPSILON 0.001
#define ADJACENT_CELLS 4
#define STRLEN 256

// ----------------------------------------------------------------------------
// Tiled Halo Cell algorithm parameters
// ----------------------------------------------------------------------------
  // Von Neumann Neighborhood
#define MASK_WIDTH 3
  // Tile size can be dynamically calculated by using tile_width = 1 - mask_width - (pow(max_shared_memory, 2) / pow(sizeof(datatype), 2))
  // max_shared_memory can be queried from the CUDA API at runtime
  // This formula is derived by solving the following equation for for tile_width:
  // max_shared_memory = (mask_width + tile_width - 1)^2 * sizeof(datatype)
  // Else, an arbitrary or estimated amount that does not surpass the GPU's capacity is chosen
#define TILE_WIDTH 30

// ----------------------------------------------------------------------------
// Read/Write access macros linearizing single/multy layer buffer 2D indices
// ----------------------------------------------------------------------------
#define SET(M, columns, i, j, value) ((M)[(((i) * (columns)) + (j))] = (value))
#define GET(M, columns, i, j) (M[(((i) * (columns)) + (j))])
#define BUF_SET(M, rows, columns, n, i, j, value) ( (M)[( ((n)*(rows)*(columns)) + ((i)*(columns)) + (j) )] = (value) )
#define BUF_GET(M, rows, columns, n, i, j) ( M[( ((n)*(rows)*(columns)) + ((i)*(columns)) + (j) )] )

// ----------------------------------------------------------------------------
// I/O functions
// ----------------------------------------------------------------------------
void readHeaderInfo(char* path, int &nrows, int &ncols, double &nodata)
{
  FILE* f;
  
  if ((f = fopen(path, "r")) == 0) {
    printf("%s configuration header file not found\n", path);
    exit(0);
  }

  //Read the header
  char str[STRLEN];
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); ncols = atoi(str);      //ncols
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); nrows = atoi(str);      //nrows
  fscanf(f,"%s",&str); fscanf(f,"%s",&str);                         //xllcorner
  fscanf(f,"%s",&str); fscanf(f,"%s",&str);                         //yllcorner
  fscanf(f,"%s",&str); fscanf(f,"%s",&str);                         //cellsize
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); nodata = atof(str);     //NODATA_value 
}

bool loadGrid2D(double *M, int rows, int columns, char *path)
{
  FILE *f = fopen(path, "r");

  if (!f) {
    printf("%s grid file not found\n", path);
    exit(0);
  }

  char str[STRLEN];
  for (int i = 0; i < rows; ++i)
    for (int j = 0; j < columns; ++j)
    {
      fscanf(f, "%s", str);
      SET(M, columns, i, j, atof(str));
    }

  fclose(f);

  return true;
}

bool saveGrid2Dr(double *M, int rows, int columns, char *path)
{
  FILE *f;
  f = fopen(path, "w");

  if (!f)
    return false;

  char str[STRLEN];
  for (int i = 0; i < rows; ++i)
  {
    for (int j = 0; j < columns; ++j)
    {
      sprintf(str, "%f ", GET(M, columns, i, j));
      fprintf(f, "%s ", str);
    }
    fprintf(f, "\n");
  }

  fclose(f);

  return true;
}

double* addLayer2D(int rows, int columns)
{
  double *tmp;
  checkError(hipMallocManaged(&tmp, sizeof(double) * rows * columns), __LINE__, "error allocating memory");

  if (!tmp)
    return NULL;
  return tmp;
}

// ----------------------------------------------------------------------------
// init kernel, called once before the simulation loop
// ----------------------------------------------------------------------------
__global__ void sciddicaTSimulationInitKernel(int r, int c, double *Sz, double *Sh)
{
  int col_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int row_idx = threadIdx.y + blockDim.y * blockIdx.y;
  int col_stride = blockDim.x * gridDim.x;
  int row_stride = blockDim.y * gridDim.y;

  double z, h;

  for (int row = row_idx + 1; row < r - 1; row += row_stride) {
    for (int col = col_idx + 1; col < c - 1; col += col_stride) {
      h = GET(Sh, c, row, col);

      if (h > 0.0) {
        z = GET(Sz, c, row, col);
        SET(Sz, c, row, col, z - h);
      }
    }
  }
}

// ----------------------------------------------------------------------------
// computing kernels, aka elementary processes in the XCA terminology
// ----------------------------------------------------------------------------
__global__ void sciddicaTResetFlowsKernel(int r, int c, double nodata, double* Sf)
{
  int col_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int row_idx = threadIdx.y + blockDim.y * blockIdx.y;
  int col_stride = blockDim.x * gridDim.x;
  int row_stride = blockDim.y * gridDim.y;

  for (int row = row_idx + 1; row < r - 1; row += row_stride) {
    for (int col = col_idx + 1; col < c - 1; col += col_stride) {
      BUF_SET(Sf, r, c, 0, row, col, 0.0);
      BUF_SET(Sf, r, c, 1, row, col, 0.0);
      BUF_SET(Sf, r, c, 2, row, col, 0.0);
      BUF_SET(Sf, r, c, 3, row, col, 0.0);
    }
  }
}

__global__ void sciddicaTFlowsComputationKernel(int r, int c, double nodata, int* Xi, int* Xj, double *Sz, double *Sh, double *Sf, double p_r, double p_epsilon)
{
  int col_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int row_idx = threadIdx.y + blockDim.y * blockIdx.y;
  int col_stride = blockDim.x * gridDim.x;
  int row_stride = blockDim.y * gridDim.y;

  bool eliminated_cells[5] = {false, false, false, false, false};
  bool again;
  int cells_count;
  double average;
  double m;
  double u[5];
  int n;
  double z, h;

  for (int row = row_idx + 1; row < r - 1; row += row_stride) {
    for (int col = col_idx + 1; col < c - 1; col += col_stride) {
      m = GET(Sh, c, row, col) - p_epsilon;
      u[0] = GET(Sz, c, row, col) + p_epsilon;
      z = GET(Sz, c, row + Xi[1], col + Xj[1]);
      h = GET(Sh, c, row + Xi[1], col + Xj[1]);
      u[1] = z + h;
      z = GET(Sz, c, row + Xi[2], col + Xj[2]);
      h = GET(Sh, c, row + Xi[2], col + Xj[2]);
      u[2] = z + h;
      z = GET(Sz, c, row + Xi[3], col + Xj[3]);
      h = GET(Sh, c, row + Xi[3], col + Xj[3]);
      u[3] = z + h;
      z = GET(Sz, c, row + Xi[4], col + Xj[4]);
      h = GET(Sh, c, row + Xi[4], col + Xj[4]);
      u[4] = z + h;

      do
      {
        again = false;
        average = m;
        cells_count = 0;

        for (n = 0; n < 5; ++n)
          if (!eliminated_cells[n])
          {
            average += u[n];
            ++cells_count;
          }

        if (cells_count != 0)
          average /= cells_count;

        for (n = 0; n < 5; ++n)
          if ((average <= u[n]) && (!eliminated_cells[n]))
          {
            eliminated_cells[n] = true;
            again = true;
          }
      } while (again);

      if (!eliminated_cells[1]) BUF_SET(Sf, r, c, 0, row, col, (average - u[1]) * p_r);
      if (!eliminated_cells[2]) BUF_SET(Sf, r, c, 1, row, col, (average - u[2]) * p_r);
      if (!eliminated_cells[3]) BUF_SET(Sf, r, c, 2, row, col, (average - u[3]) * p_r);
      if (!eliminated_cells[4]) BUF_SET(Sf, r, c, 3, row, col, (average - u[4]) * p_r);
    }
  }
}

__global__ void sciddicaTFlowsComputationCachingKernel(int r, int c, double nodata, int* Xi, int* Xj, double *Sz, double *Sh, double *Sf, double p_r, double p_epsilon)
{
  int col_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int row_idx = threadIdx.y + blockDim.y * blockIdx.y;

  bool eliminated_cells[5] = {false, false, false, false, false};
  bool again;
  int cells_count;
  double average;
  double m;
  double u[5];
  int n;
  double z, h;

  __shared__ double Sz_ds[TILE_WIDTH][TILE_WIDTH];
  __shared__ double Sh_ds[TILE_WIDTH][TILE_WIDTH];

  Sz_ds[threadIdx.y][threadIdx.x] = GET(Sz, c, row_idx, col_idx);
  Sh_ds[threadIdx.y][threadIdx.x] = GET(Sh, c, row_idx, col_idx);
  __syncthreads();

  int tile_start_x = blockIdx.x * blockDim.x;
  int next_tile_start_x = ((blockIdx.x + 1) * blockDim.x);
  int tile_start_y = blockIdx.y * blockDim.y;
  int next_tile_start_y = ((blockIdx.y + 1) * blockDim.y);

  if (row_idx > 0 && row_idx < r - 1 && col_idx > 0 && col_idx < c - 1) {
    m = Sh_ds[threadIdx.y][threadIdx.x] - p_epsilon;
    u[0] = Sz_ds[threadIdx.y][threadIdx.x] + p_epsilon;

    for(int cnt = 0; cnt <= MASK_WIDTH; ++cnt) {
      int n_index_x = row_idx + Xi[cnt + 1];
      int n_index_y = col_idx + Xj[cnt + 1];
      if((n_index_x >= 0) && (n_index_x < c) && (n_index_y >= 0) && (n_index_y < r)) {
        if((n_index_x >= tile_start_x) && (n_index_x < next_tile_start_x) && (n_index_y >= tile_start_y) && (n_index_y < next_tile_start_y)) {
          z = Sz_ds[threadIdx.y + Xi[cnt + 1]][threadIdx.x + Xj[cnt + 1]];
          h = Sh_ds[threadIdx.y + Xi[cnt + 1]][threadIdx.x + Xj[cnt + 1]];
        }
        else {  // try to get a L2 cache hit (best case, otherwise global memory in DRAM has to be accessed)
          z = GET(Sz, c, n_index_y, n_index_x);
          h = GET(Sh, c, n_index_y, n_index_x);
        }
      }
      u[cnt + 1] = z + h;
    }

    do
    {
      again = false;
      average = m;
      cells_count = 0;

      for (n = 0; n < 5; ++n)
        if (!eliminated_cells[n])
        {
          average += u[n];
          ++cells_count;
        }

      if (cells_count != 0)
        average /= cells_count;

      for (n = 0; n < 5; ++n)
        if ((average <= u[n]) && (!eliminated_cells[n]))
        {
          eliminated_cells[n] = true;
          again = true;
        }
    } while (again);

    for(int cnt = 0; cnt <= MASK_WIDTH; ++cnt) {
      if (!eliminated_cells[cnt + 1]) BUF_SET(Sf, r, c, cnt, row_idx, col_idx, (average - u[cnt + 1]) * p_r);
    }
  }
}

__global__ void sciddicaTWidthUpdateKernel(int r, int c, double nodata, int* Xi, int* Xj, double *Sz, double *Sh, double *Sf)
{
  int col_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int row_idx = threadIdx.y + blockDim.y * blockIdx.y;
  int col_stride = blockDim.x * gridDim.x;
  int row_stride = blockDim.y * gridDim.y;

  double h_next;

  for (int row = row_idx + 1; row < r - 1; row += row_stride) {
    for (int col = col_idx + 1; col < c - 1; col += col_stride) {
      h_next = GET(Sh, c, row, col);
      h_next += BUF_GET(Sf, r, c, 3, row+Xi[1], col+Xj[1]) - BUF_GET(Sf, r, c, 0, row, col);
      h_next += BUF_GET(Sf, r, c, 2, row+Xi[2], col+Xj[2]) - BUF_GET(Sf, r, c, 1, row, col);
      h_next += BUF_GET(Sf, r, c, 1, row+Xi[3], col+Xj[3]) - BUF_GET(Sf, r, c, 2, row, col);
      h_next += BUF_GET(Sf, r, c, 0, row+Xi[4], col+Xj[4]) - BUF_GET(Sf, r, c, 3, row, col);

      SET(Sh, c, row, col, h_next);
    }
  }
}

__global__ void sciddicaTWidthUpdateCachingKernel(int r, int c, double nodata, int* Xi, int* Xj, double *Sz, double *Sh, double *Sf)
{
  int col_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int row_idx = threadIdx.y + blockDim.y * blockIdx.y;

  double h_next;
  
  __shared__ double Sf_ds[TILE_WIDTH * ADJACENT_CELLS][TILE_WIDTH];

  for(int cnt = 0; cnt < 4; ++cnt) {
    Sf_ds[threadIdx.y + cnt * TILE_WIDTH][threadIdx.x] = BUF_GET(Sf, r, c, cnt, row_idx, col_idx);
  }
  __syncthreads();

  int tile_start_x = blockIdx.x * blockDim.x;
  int next_tile_start_x = ((blockIdx.x + 1) * blockDim.x);
  int tile_start_y = blockIdx.y * blockDim.y;
  int next_tile_start_y = ((blockIdx.y + 1) * blockDim.y);

  if(col_idx > 0 && col_idx < c - 1 && row_idx > 0 && row_idx < r - 1) {
    h_next = GET(Sh, c, row_idx, col_idx);

    for(int cnt = 0; cnt <= MASK_WIDTH; ++cnt) {
      int n_index_x = col_idx + Xj[cnt + 1];
      int n_index_y = row_idx + Xi[cnt + 1];
      if((n_index_x >= 0) && (n_index_x < c) && (n_index_y >= 0) && (n_index_y < r)) {
        if((n_index_x >= tile_start_x) && (n_index_x < next_tile_start_x) && (n_index_y >= tile_start_y) && (n_index_y < next_tile_start_y)) {
          h_next += Sf_ds[threadIdx.y + Xi[cnt + 1] + (MASK_WIDTH - cnt) * TILE_WIDTH][threadIdx.x + Xj[cnt + 1]]
                    - Sf_ds[threadIdx.y + cnt * TILE_WIDTH][threadIdx.x];
        }
        else {  // try to get a L2 cache hit (best case, otherwise global memory in DRAM has to be accessed)
          h_next += BUF_GET(Sf, r, c, (MASK_WIDTH - cnt), n_index_y, n_index_x)
                    - BUF_GET(Sf, r, c, cnt, row_idx, col_idx);
        }
      }
    }

    SET(Sh, c, row_idx, col_idx, h_next);
  }
}

// This kernel benefits from a tiled implementation
__global__ void sciddicaTFlowsComputationKernelVerified(int r, int c, double nodata, int *Xi, int *Xj, double *Sz, double *Sh, double *Sf, double p_r, double p_epsilon)
{
  int col_index = threadIdx.x + blockDim.x * blockIdx.x;
  int row_index = threadIdx.y + blockDim.y * blockIdx.y;

  bool eliminated_cells[5] = {false, false, false, false, false};
  bool again;
  int cells_count;
  double average;
  double m;
  double u[5];
  int n;
  double z = 0, h = 0;

  __shared__ double Sz_ds[TILE_WIDTH][TILE_WIDTH];
  __shared__ double Sh_ds[TILE_WIDTH][TILE_WIDTH];

  Sz_ds[threadIdx.y][threadIdx.x] = GET(Sz, c, row_index, col_index);
  Sh_ds[threadIdx.y][threadIdx.x] = GET(Sh, c, row_index, col_index);
  __syncthreads();

  int tile_start_x = blockIdx.x * blockDim.x;
  int next_tile_start_x = ((blockIdx.x + 1) * blockDim.x);
  int tile_start_y = blockIdx.y * blockDim.y;
  int next_tile_start_y = ((blockIdx.y + 1) * blockDim.y);

  if (row_index > 0 && row_index < r - 1 && col_index > 0 && col_index < c - 1)
  {
    m = Sh_ds[threadIdx.y][threadIdx.x] - p_epsilon;
    u[0] = Sz_ds[threadIdx.y][threadIdx.x] + p_epsilon;

    int index_x;
    int index_y;

    for (int tmp = 0; tmp <= MASK_WIDTH; tmp++)
    {
      index_y = row_index + Xi[tmp + 1];
      index_x = col_index + Xj[tmp + 1];

      if ((index_x >= 0) && (index_x < c) && (index_y >= 0) && (index_y < r))
      {
        if ((index_x >= tile_start_x) && (index_x < next_tile_start_x) && (index_y >= tile_start_y) && (index_y < next_tile_start_y))
        {
          z = Sz_ds[threadIdx.y + Xi[tmp + 1]][threadIdx.x + Xj[tmp + 1]];
          h = Sh_ds[threadIdx.y + Xi[tmp + 1]][threadIdx.x + Xj[tmp + 1]];
        }
        else
        {
          z = GET(Sz, c, index_y, index_x);
          h = GET(Sh, c, index_y, index_x);
        }
        u[tmp + 1] = z + h;
      }
    }

    do
    {
      again = false;
      average = m;
      cells_count = 0;

      for (n = 0; n < 5; n++)
        if (!eliminated_cells[n])
        {
          average += u[n];
          cells_count++;
        }

      if (cells_count != 0)
        average /= cells_count;

      for (n = 0; n < 5; n++)
        if ((average <= u[n]) && (!eliminated_cells[n]))
        {
          eliminated_cells[n] = true;
          again = true;
        }
    } while (again);

    if (!eliminated_cells[1])
      BUF_SET(Sf, r, c, 0, row_index, col_index, (average - u[1]) * p_r);
    if (!eliminated_cells[2])
      BUF_SET(Sf, r, c, 1, row_index, col_index, (average - u[2]) * p_r);
    if (!eliminated_cells[3])
      BUF_SET(Sf, r, c, 2, row_index, col_index, (average - u[3]) * p_r);
    if (!eliminated_cells[4])
      BUF_SET(Sf, r, c, 3, row_index, col_index, (average - u[4]) * p_r);
  }
}

// This kernel benefits from a tiled implementation
__global__ void sciddicaTWidthUpdateKernelVerified(int r, int c, double nodata, int *Xi, int *Xj, double *Sz, double *Sh, double *Sf)
{
  int col_index = threadIdx.x + blockDim.x * blockIdx.x;
  int row_index = threadIdx.y + blockDim.y * blockIdx.y;

  double h_next;

  __shared__ double Sf_ds[TILE_WIDTH * ADJACENT_CELLS][TILE_WIDTH];

  Sf_ds[threadIdx.y][threadIdx.x] = BUF_GET(Sf, r, c, 0, row_index, col_index);
  Sf_ds[threadIdx.y + TILE_WIDTH][threadIdx.x] = BUF_GET(Sf, r, c, 1, row_index, col_index);
  Sf_ds[threadIdx.y + TILE_WIDTH * 2][threadIdx.x] = BUF_GET(Sf, r, c, 2, row_index, col_index);
  Sf_ds[threadIdx.y + TILE_WIDTH * 3][threadIdx.x] = BUF_GET(Sf, r, c, 3, row_index, col_index);
  __syncthreads();

  int tile_start_x = blockIdx.x * blockDim.x;
  int next_tile_start_x = ((blockIdx.x + 1) * blockDim.x);
  int tile_start_y = blockIdx.y * blockDim.y;
  int next_tile_start_y = ((blockIdx.y + 1) * blockDim.y);

  if (row_index > 0 && row_index < r - 1 && col_index > 0 && col_index < c - 1)
  {
    h_next = GET(Sh, c, row_index, col_index);

    for (int tmp = 0; tmp <= MASK_WIDTH; ++tmp)
    {
      int n_index_x = col_index + Xj[tmp + 1];
      int n_index_y = row_index + Xi[tmp + 1];
      if ((n_index_x >= 0) && (n_index_x < c) && (n_index_y >= 0) && (n_index_y < r))
      {
        if ((n_index_x >= tile_start_x) && (n_index_x < next_tile_start_x) && (n_index_y >= tile_start_y) && (n_index_y < next_tile_start_y))
        {
          h_next += Sf_ds[threadIdx.y + TILE_WIDTH * (MASK_WIDTH - tmp) + Xi[tmp + 1]][threadIdx.x + Xj[tmp + 1]] - Sf_ds[threadIdx.y + TILE_WIDTH * tmp][threadIdx.x];
        }
        else
        {
          h_next += BUF_GET(Sf, r, c, (MASK_WIDTH - tmp), n_index_y, n_index_x) - BUF_GET(Sf, r, c, tmp, row_index, col_index);
        }
      }
    }
    SET(Sh, c, row_index, col_index, h_next);
  }
}

// ----------------------------------------------------------------------------
// Function main()
// ----------------------------------------------------------------------------
int main(int argc, char **argv)
{
  int rows, cols;
  double nodata;
  readHeaderInfo(argv[HEADER_PATH_ID], rows, cols, nodata);

  int r = rows;                     // r: grid rows
  int c = cols;                     // c: grid columns
  // int i_start = 1, i_end = r-1;   // [i_start,i_end[: kernel application range along rows
  // int j_start = 1, j_end = c-1;   // [i_start,i_end[: kernel application range along columns
  double *Sz;                       // Sz: substate (grid) containing cells' altitude a.s.l.
  double *Sh;                       // Sh: substate (grid) containing cells' flow thickness
  double *Sf;                       // Sf: 4 substates containing the flows towards the 4 neighbors
  int* Xi;                          // Xj: von Neuman neighborhood row coordinates (see below)
  int* Xj;                          // Xj: von Neuman neighborhood col coordinates (see below)
  double p_r = P_R;                 // p_r: minimization algorithm outflows dumping factor
  double p_epsilon = P_EPSILON;     // p_epsilon: frictional parameter threshold
  int steps = atoi(argv[STEPS_ID]); //steps: simulation steps

  // The adopted von Neuman neighborhood
  // Format: flow_index:cell_label:(row_index,col_index)
  //
  //   cell_label in [0,1,2,3,4]: label assigned to each cell in the neighborhood
  //   flow_index in   [0,1,2,3]: outgoing flow indices in Sf from cell 0 to the others
  //       (row_index,col_index): 2D relative indices of the cells
  //
  //               |0:1:(-1, 0)|
  //   |1:2:( 0,-1)| :0:( 0, 0)|2:3:( 0, 1)|
  //               |3:4:( 1, 0)|
  //
  //

  // printf("Allocating memory...\n");
  Sz = addLayer2D(r, c);                  // Allocates the Sz substate grid
  Sh = addLayer2D(r, c);                  // Allocates the Sh substate grid
  Sf = addLayer2D(ADJACENT_CELLS * r, c); // Allocates the Sf substates grid, having one layer for each adjacent cell
  checkError(hipMallocManaged(&Xi, sizeof(int) * 5), __LINE__, "error allocating memory for Xi");
  Xi[0] = 0; Xi[1] = -1; Xi[2] = 0;  Xi[3] = 0; Xi[4] = 1;
  checkError(hipMallocManaged(&Xj, sizeof(int) * 5), __LINE__, "error allocating memory for Xj");
  Xj[0] = 0; Xj[1] = 0;  Xj[2] = -1; Xj[3] = 1; Xj[4] = 0;

  // printf("Loading data from file...\n");
  loadGrid2D(Sz, r, c, argv[DEM_PATH_ID]);    // Load Sz from file
  loadGrid2D(Sh, r, c, argv[SOURCE_PATH_ID]); // Load Sh from file

  int n = rows * cols;
  int dim_x = 32;
  int dim_y = 32;
  dim3 block_size(dim_x, dim_y, 1);
  dim3 grid_size(ceil(sqrt(n / (dim_x * dim_y))), ceil(sqrt(n / (dim_x * dim_y))), 1);

  // printf("Problem size is %d elements\n", n);
  // printf("Block dimensions are %d, %d, %d\n", block_size.x, block_size.y, block_size.z);
  // printf("Grid dimensions are %d, %d, %d\n", grid_size.x, grid_size.y, grid_size.z);
  // printf("Total grid threads are: %d\n", block_size.x * block_size.y * grid_size.x * grid_size.y);

  dim3 tiled_block_size(TILE_WIDTH, TILE_WIDTH, 1);
  dim3 tiled_grid_size(ceil(sqrt(n / (TILE_WIDTH * TILE_WIDTH))), ceil(sqrt(n / (TILE_WIDTH * TILE_WIDTH))), 1);

  printf("\n");
  printf("Mask width is %d\n", MASK_WIDTH);
  printf("Tile width is %d\n", TILE_WIDTH);
  printf("Problem size is %d elements\n", n);
  printf("Tiled block dimensions are %d, %d, %d\n", tiled_block_size.x, tiled_block_size.y, tiled_block_size.z);
  printf("Tiled grid dimensions are %d, %d, %d\n", tiled_grid_size.x, tiled_grid_size.y, tiled_grid_size.z);
  printf("Total blocks in tiled grid are: %d\n", tiled_grid_size.x * tiled_grid_size.y * tiled_grid_size.z);
  printf("Total tiled grid threads are: %d\n", tiled_block_size.x * tiled_block_size.y * tiled_block_size.z * tiled_grid_size.x * tiled_grid_size.y * tiled_grid_size.z);
  printf("Threads only involved in output: %d\n", TILE_WIDTH * TILE_WIDTH * tiled_grid_size.x * tiled_grid_size.y * tiled_grid_size.z);
  printf("One double precision buffer requires %lld bytes of shared memory\n", TILE_WIDTH * TILE_WIDTH * sizeof(double));
  printf("\n");

  //printf("Initializing...\n");
  sciddicaTSimulationInitKernel<<<grid_size, block_size>>>(r, c, Sz, Sh);
  checkError(__LINE__, "error executing sciddicaTSimulationInitKernel");
  checkError(hipDeviceSynchronize(), __LINE__, "error syncing after sciddicaTSimulationInitKernel");

  // int loops = 100;  // TEST
  printf("Running the simulation for %d steps...\n", steps);
  // printf("... and %d times, determining the best time.\n", loops);
  // double best_time = 0.0;
  // for(int loop = 0; loop < loops; ++loop) {
    util::Timer cl_timer;
    for (int s = 0; s < steps; ++s) {
      //printf("step %d\n", s+1);

      sciddicaTResetFlowsKernel<<<grid_size, block_size>>>(r, c, nodata, Sf);
      checkError(__LINE__, "error executing sciddicaTSimulationInitKernel");
      checkError(hipDeviceSynchronize(), __LINE__, "error syncing after sciddicaTResetFlowsKernel");

      // sciddicaTFlowsComputationKernel<<<grid_size, block_size>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf, p_r, p_epsilon);
      // checkError(__LINE__, "error executing sciddicaTFlowsComputationKernel");
      // checkError(hipDeviceSynchronize(), __LINE__, "error syncing after sciddicaTFlowsComputationKernel");

      sciddicaTFlowsComputationKernelVerified<<<tiled_grid_size, tiled_block_size>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf, p_r, p_epsilon);
      checkError(__LINE__, "error executing sciddicaTFlowsComputationCachingKernel");
      checkError(hipDeviceSynchronize(), __LINE__, "error syncing after sciddicaTFlowsComputationCachingKernel");

      // sciddicaTWidthUpdateKernel<<<grid_size, block_size>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf);
      // checkError(__LINE__, "error executing sciddicaTWidthUpdateKernel");
      // checkError(hipDeviceSynchronize(), __LINE__, "error syncing after sciddicaTWidthUpdateKernel");

      sciddicaTWidthUpdateKernelVerified<<<tiled_grid_size, tiled_block_size>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf);
      checkError(__LINE__, "error executing sciddicaTWidthUpdateCachingKernel");
      checkError(hipDeviceSynchronize(), __LINE__, "error syncing after sciddicaTWidthUpdateCachingKernel");
    }
    double cl_time = static_cast<double>(cl_timer.getTimeMilliseconds()) / 1000.0;
    // printf("[%d] ", loop);
    printf("Elapsed time: %lf [s]\n", cl_time);
    // if(cl_time < best_time || loop == 0) {
    //   best_time = cl_time;
    // }
  // }
  // printf("Best time: %lf [s]\n", best_time);

  saveGrid2Dr(Sh, r, c, argv[OUTPUT_PATH_ID]);

  //printf("Releasing memory...\n");
  checkError(hipFree(Sz), __LINE__, "error deallocating memory for Sz");
  checkError(hipFree(Sh), __LINE__, "error deallocating memory for Sh");
  checkError(hipFree(Sf), __LINE__, "error deallocating memory for Sf");
  checkError(hipFree(Xi), __LINE__, "error deallocating memory for Xi");
  checkError(hipFree(Xj), __LINE__, "error deallocating memory for Xj");

  return 0;
}
