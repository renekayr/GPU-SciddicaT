#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "util.hpp"

// ----------------------------------------------------------------------------
// I/O parameters used to index argv[]
// ----------------------------------------------------------------------------
#define HEADER_PATH_ID 1
#define DEM_PATH_ID 2
#define SOURCE_PATH_ID 3
#define OUTPUT_PATH_ID 4
#define STEPS_ID 5

// ----------------------------------------------------------------------------
// Simulation parameters
// ----------------------------------------------------------------------------
#define P_R 0.5
#define P_EPSILON 0.001
#define ADJACENT_CELLS 4
#define STRLEN 256

// ----------------------------------------------------------------------------
// Read/Write access macros linearizing single/multy layer buffer 2D indices
// ----------------------------------------------------------------------------
#define SET(M, columns, i, j, value) ((M)[(((i) * (columns)) + (j))] = (value))
#define GET(M, columns, i, j) (M[(((i) * (columns)) + (j))])
#define BUF_SET(M, rows, columns, n, i, j, value) ( (M)[( ((n)*(rows)*(columns)) + ((i)*(columns)) + (j) )] = (value) )
#define BUF_GET(M, rows, columns, n, i, j) ( M[( ((n)*(rows)*(columns)) + ((i)*(columns)) + (j) )] )

// ----------------------------------------------------------------------------
// I/O functions
// ----------------------------------------------------------------------------
void readHeaderInfo(char* path, int &nrows, int &ncols, double &nodata)
{
  FILE* f;
  
  if ((f = fopen(path, "r")) == 0) {
    printf("%s configuration header file not found\n", path);
    exit(0);
  }

  //Read the header
  char str[STRLEN];
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); ncols = atoi(str);      //ncols
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); nrows = atoi(str);      //nrows
  fscanf(f,"%s",&str); fscanf(f,"%s",&str);                         //xllcorner
  fscanf(f,"%s",&str); fscanf(f,"%s",&str);                         //yllcorner
  fscanf(f,"%s",&str); fscanf(f,"%s",&str);                         //cellsize
  fscanf(f,"%s",&str); fscanf(f,"%s",&str); nodata = atof(str);     //NODATA_value 
}

bool loadGrid2D(double *M, int rows, int columns, char *path)
{
  FILE *f = fopen(path, "r");

  if (!f) {
    printf("%s grid file not found\n", path);
    exit(0);
  }

  char str[STRLEN];
  for (int i = 0; i < rows; ++i)
    for (int j = 0; j < columns; ++j)
    {
      fscanf(f, "%s", str);
      SET(M, columns, i, j, atof(str));
    }

  fclose(f);

  return true;
}

bool saveGrid2Dr(double *M, int rows, int columns, char *path)
{
  FILE *f;
  f = fopen(path, "w");

  if (!f)
    return false;

  char str[STRLEN];
  for (int i = 0; i < rows; ++i)
  {
    for (int j = 0; j < columns; ++j)
    {
      sprintf(str, "%f ", GET(M, columns, i, j));
      fprintf(f, "%s ", str);
    }
    fprintf(f, "\n");
  }

  fclose(f);

  return true;
}

double* addLayer2D(int rows, int columns)
{
  double *tmp;
  checkError(hipMallocManaged(&tmp, sizeof(double) * rows * columns), __LINE__, "error allocating memory");

  if (!tmp)
    return NULL;
  return tmp;
}

// ----------------------------------------------------------------------------
// init kernel, called once before the simulation loop
// ----------------------------------------------------------------------------
__global__ void sciddicaTSimulationInitKernel(int r, int c, double *Sz, double *Sh)
{
  int col_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int row_idx = threadIdx.y + blockDim.y * blockIdx.y;
  int col_stride = blockDim.x * gridDim.x;
  int row_stride = blockDim.y * gridDim.y;

  double z, h;

  for (int row = row_idx + 1; row < r - 1; row += row_stride) {
    for (int col = col_idx + 1; col < c - 1; col += col_stride) {
      h = GET(Sh, c, row, col);

      if (h > 0.0) {
        z = GET(Sz, c, row, col);
        SET(Sz, c, row, col, z - h);
      }
    }
  }
}

// ----------------------------------------------------------------------------
// computing kernels, aka elementary processes in the XCA terminology
// ----------------------------------------------------------------------------
__global__ void sciddicaTResetFlowsKernel(int r, int c, double nodata, double* Sf)
{
  int col_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int row_idx = threadIdx.y + blockDim.y * blockIdx.y;
  int col_stride = blockDim.x * gridDim.x;
  int row_stride = blockDim.y * gridDim.y;

  for (int row = row_idx + 1; row < r - 1; row += row_stride) {
    for (int col = col_idx + 1; col < c - 1; col += col_stride) {
      BUF_SET(Sf, r, c, 0, row, col, 0.0);
      BUF_SET(Sf, r, c, 1, row, col, 0.0);
      BUF_SET(Sf, r, c, 2, row, col, 0.0);
      BUF_SET(Sf, r, c, 3, row, col, 0.0);
    }
  }
}

__global__ void sciddicaTFlowsComputationKernel(int r, int c, double nodata, int* Xi, int* Xj, double *Sz, double *Sh, double *Sf, double p_r, double p_epsilon)
{
  int col_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int row_idx = threadIdx.y + blockDim.y * blockIdx.y;
  int col_stride = blockDim.x * gridDim.x;
  int row_stride = blockDim.y * gridDim.y;

  bool eliminated_cells[5] = {false, false, false, false, false};
  bool again;
  int cells_count;
  double average;
  double m;
  double u[5];
  int n;
  double z, h;

  for (int row = row_idx + 1; row < r - 1; row += row_stride) {
    for (int col = col_idx + 1; col < c - 1; col += col_stride) {
      m = GET(Sh, c, row, col) - p_epsilon;
      u[0] = GET(Sz, c, row, col) + p_epsilon;
      z = GET(Sz, c, row + Xi[1], col + Xj[1]);
      h = GET(Sh, c, row + Xi[1], col + Xj[1]);
      u[1] = z + h;
      z = GET(Sz, c, row + Xi[2], col + Xj[2]);
      h = GET(Sh, c, row + Xi[2], col + Xj[2]);
      u[2] = z + h;
      z = GET(Sz, c, row + Xi[3], col + Xj[3]);
      h = GET(Sh, c, row + Xi[3], col + Xj[3]);
      u[3] = z + h;
      z = GET(Sz, c, row + Xi[4], col + Xj[4]);
      h = GET(Sh, c, row + Xi[4], col + Xj[4]);
      u[4] = z + h;

      do
      {
        again = false;
        average = m;
        cells_count = 0;

        for (n = 0; n < 5; ++n)
          if (!eliminated_cells[n])
          {
            average += u[n];
            ++cells_count;
          }

        if (cells_count != 0)
          average /= cells_count;

        for (n = 0; n < 5; ++n)
          if ((average <= u[n]) && (!eliminated_cells[n]))
          {
            eliminated_cells[n] = true;
            again = true;
          }
      } while (again);

      if (!eliminated_cells[1]) BUF_SET(Sf, r, c, 0, row, col, (average - u[1]) * p_r);
      if (!eliminated_cells[2]) BUF_SET(Sf, r, c, 1, row, col, (average - u[2]) * p_r);
      if (!eliminated_cells[3]) BUF_SET(Sf, r, c, 2, row, col, (average - u[3]) * p_r);
      if (!eliminated_cells[4]) BUF_SET(Sf, r, c, 3, row, col, (average - u[4]) * p_r);
    }
  }
}

__global__ void sciddicaTWidthUpdateKernel(int r, int c, double nodata, int* Xi, int* Xj, double *Sz, double *Sh, double *Sf)
{
  int col_idx = threadIdx.x + blockDim.x * blockIdx.x;
  int row_idx = threadIdx.y + blockDim.y * blockIdx.y;
  int col_stride = blockDim.x * gridDim.x;
  int row_stride = blockDim.y * gridDim.y;

  double h_next;

  for (int row = row_idx + 1; row < r - 1; row += row_stride) {
    for (int col = col_idx + 1; col < c - 1; col += col_stride) {
      h_next = GET(Sh, c, row, col);
      h_next += BUF_GET(Sf, r, c, 3, row+Xi[1], col+Xj[1]) - BUF_GET(Sf, r, c, 0, row, col);
      h_next += BUF_GET(Sf, r, c, 2, row+Xi[2], col+Xj[2]) - BUF_GET(Sf, r, c, 1, row, col);
      h_next += BUF_GET(Sf, r, c, 1, row+Xi[3], col+Xj[3]) - BUF_GET(Sf, r, c, 2, row, col);
      h_next += BUF_GET(Sf, r, c, 0, row+Xi[4], col+Xj[4]) - BUF_GET(Sf, r, c, 3, row, col);

      SET(Sh, c, row, col, h_next);
    }
  }
}

// ----------------------------------------------------------------------------
// Function main()
// ----------------------------------------------------------------------------
int main(int argc, char **argv)
{
  int rows, cols;
  double nodata;
  readHeaderInfo(argv[HEADER_PATH_ID], rows, cols, nodata);

  int r = rows;                     // r: grid rows
  int c = cols;                     // c: grid columns
  //int i_start = 1, i_end = r-1;   // [i_start,i_end[: kernel application range along rows
  //int j_start = 1, j_end = c-1;   // [i_start,i_end[: kernel application range along columns
  double *Sz;                       // Sz: substate (grid) containing cells' altitude a.s.l.
  double *Sh;                       // Sh: substate (grid) containing cells' flow thickness
  double *Sf;                       // Sf: 4 substates containing the flows towards the 4 neighbors
  int* Xi;                          // Xj: von Neuman neighborhood row coordinates (see below)
  int* Xj;                          // Xj: von Neuman neighborhood col coordinates (see below)
  double p_r = P_R;                 // p_r: minimization algorithm outflows dumping factor
  double p_epsilon = P_EPSILON;     // p_epsilon: frictional parameter threshold
  int steps = atoi(argv[STEPS_ID]); //steps: simulation steps

  // The adopted von Neuman neighborhood
  // Format: flow_index:cell_label:(row_index,col_index)
  //
  //   cell_label in [0,1,2,3,4]: label assigned to each cell in the neighborhood
  //   flow_index in   [0,1,2,3]: outgoing flow indices in Sf from cell 0 to the others
  //       (row_index,col_index): 2D relative indices of the cells
  //
  //               |0:1:(-1, 0)|
  //   |1:2:( 0,-1)| :0:( 0, 0)|2:3:( 0, 1)|
  //               |3:4:( 1, 0)|
  //
  //

  //printf("Allocating memory...\n");
  Sz = addLayer2D(r, c);                  // Allocates the Sz substate grid
  Sh = addLayer2D(r, c);                  // Allocates the Sh substate grid
  Sf = addLayer2D(ADJACENT_CELLS * r, c); // Allocates the Sf substates grid, having one layer for each adjacent cell
  checkError(hipMallocManaged(&Xi, sizeof(int) * 5), __LINE__, "error allocating memory for Xi");
  Xi[0] = 0; Xi[1] = -1; Xi[2] = 0;  Xi[3] = 0; Xi[4] = 1;
  checkError(hipMallocManaged(&Xj, sizeof(int) * 5), __LINE__, "error allocating memory for Xj");
  Xj[0] = 0; Xj[1] = 0;  Xj[2] = -1; Xj[3] = 1; Xj[4] = 0;

  //printf("Loading data from file...\n");
  loadGrid2D(Sz, r, c, argv[DEM_PATH_ID]);    // Load Sz from file
  loadGrid2D(Sh, r, c, argv[SOURCE_PATH_ID]); // Load Sh from file

  int n = rows * cols;
  int dim_x = 32;
  int dim_y = 32;
  dim3 block_size(dim_x, dim_y, 1);
  dim3 grid_size(ceil(sqrt(n / (dim_x * dim_y))), ceil(sqrt(n / (dim_x * dim_y))), 1);

  // printf("Problem size is %d elements\n", n);
  // printf("Block dimensions are %d, %d, %d\n", block_size.x, block_size.y, block_size.z);
  // printf("Grid dimensions are %d, %d, %d\n", grid_size.x, grid_size.y, grid_size.z);
  // printf("Total grid threads are: %d\n", block_size.x * block_size.y * grid_size.x * grid_size.y);

  //printf("Initializing...\n");
  sciddicaTSimulationInitKernel<<<grid_size, block_size>>>(r, c, Sz, Sh);
  checkError(__LINE__, "error executing sciddicaTSimulationInitKernel");
  checkError(hipDeviceSynchronize(), __LINE__, "error syncing after sciddicaTSimulationInitKernel");

  int loops = 100;  // TEST
  printf("Running the simulation for %d steps...\n", steps);
  printf("... and %d times, determining the best time.\n", loops);
  double best_time = 0.0;
  for(int loop = 0; loop < loops; ++loop) {
    util::Timer cl_timer;
    for (int s = 0; s < steps; ++s) {
      //printf("step %d\n", s+1);

      sciddicaTResetFlowsKernel<<<grid_size, block_size>>>(r, c, nodata, Sf);
      checkError(__LINE__, "error executing sciddicaTSimulationInitKernel");
      checkError(hipDeviceSynchronize(), __LINE__, "error syncing after sciddicaTResetFlowsKernel");

      sciddicaTFlowsComputationKernel<<<grid_size, block_size>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf, p_r, p_epsilon);
      checkError(__LINE__, "error executing sciddicaTFlowsComputationKernel");
      checkError(hipDeviceSynchronize(), __LINE__, "error syncing after sciddicaTFlowsComputationKernel");
      
      sciddicaTWidthUpdateKernel<<<grid_size, block_size>>>(r, c, nodata, Xi, Xj, Sz, Sh, Sf);
      checkError(__LINE__, "error executing sciddicaTWidthUpdateKernel");
      checkError(hipDeviceSynchronize(), __LINE__, "error syncing after sciddicaTWidthUpdateKernel");
    }
    double cl_time = static_cast<double>(cl_timer.getTimeMilliseconds()) / 1000.0;
    printf("[%d] ", loop);
    printf("Elapsed time: %lf [s]\n", cl_time);
    if(cl_time < best_time || loop == 0) {
      best_time = cl_time;
    }
  }
  printf("Best time: %lf [s]\n", best_time);

  saveGrid2Dr(Sh, r, c, argv[OUTPUT_PATH_ID]);

  //printf("Releasing memory...\n");
  checkError(hipFree(Sz), __LINE__, "error deallocating memory for Sz");
  checkError(hipFree(Sh), __LINE__, "error deallocating memory for Sh");
  checkError(hipFree(Sf), __LINE__, "error deallocating memory for Sf");
  checkError(hipFree(Xi), __LINE__, "error deallocating memory for Xi");
  checkError(hipFree(Xj), __LINE__, "error deallocating memory for Xj");

  return 0;
}
